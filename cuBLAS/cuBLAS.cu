#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>


int main(int argc, char **argv)
{
    if (argc != 2)
    {
        printf("Usage: %s <matrix_dimension>\n", argv[0]);
        return 1;
    }

    // Matrix dimension
    int n = atoi(argv[1);

    // Allocate memory for matrices on the host
    float *h_A = (float *)malloc(n * n * sizeof(float));
    float *h_B = (float *)malloc(n * n * sizeof(float));
    float *h_C = (float *)malloc(n * n * sizeof(float));

    // Initialize matrices A and B with random values
    for(int i = 0; i < n * n; i++)
    {
        h_A[i] = rand() % 10 + 1;
        h_B[i] = rand() % 10 + 1;
    }

    // Allocate memory for matrices on the device
    float *d_A, *d_B, *d_C;
    hipMalloc((void **)&d_A, n * n * sizeof(float));
    hipMalloc((void **)&d_B, n * n * sizeof(float));
    hipMalloc((void **)&d_C, n * n * sizeof(float));

    // Copy matrices from host to device
    hipMemcpy(d_A, h_A, n * n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, n * n * sizeof(float), hipMemcpyHostToDevice);

    // Create cuBLAS handle
    hipblasHandle_t handle;
    hipblasCreate(&handle);

    // Perform matrix multiplication using cuBLAS
    const float alpha = 1.0f;
    const float beta = 0.0f;
    hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, n, n, n, &alpha, d_A, n, d_B, n, &beta, d_C, n);

    // Copy the result matrix from device to host
    hipMemcpy(h_C, d_C, n * n * sizeof(float), hipMemcpyDeviceToHost);

    // Print the result matrix (C)
    printf("Result matrix (C):\n");
    for (int i = 0; i < n; i++)
    {
        for (int j = 0; j < n; j++)
        {
            printf("%f\t", h_C[i * n + j]);
        }
        printf("\n");
    }

    // Cleanup
    free(h_A);
    free(h_B);
    free(h_C);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    hipblasDestroy(handle);

    return 0;
}
